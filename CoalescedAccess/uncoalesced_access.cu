
#include <hip/hip_runtime.h>
__global__ void AccessKernel(float* in, float* out, int64_t elem_cnt){
    const int32_t idx = threadIdx.x; 
    out[idx*2] = in[idx*2]; 
    out[idx*2+1] = in[idx*2+1]; 
}

int main(){
    int64_t elem_cnt = 64; 
    float* device_in; 
    float* device_out;
    hipMalloc(&device_in, elem_cnt * sizeof(float)); 
    hipMalloc(&device_out, elem_cnt * sizeof(float)); 

    AccessKernel<<<1, 32>>>(device_in, device_out, elem_cnt); 

    hipFree(device_in); 
    hipFree(device_out); 

    return 0; 
}